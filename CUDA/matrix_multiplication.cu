#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <winsock.h>

inline void CHECK(const hipError_t error)
{
    if(error != hipSuccess) {
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));
        exit(1);
    }
}

int gettimeofday (struct timeval *tv, void* tz)
{
	union {
		long long int ns100; /*time since 1 Jan 1601 in 100ns units */
		FILETIME ft;
	} now;

	GetSystemTimeAsFileTime (&now.ft);
	tv->tv_usec = (long long int) ((now.ns100 / 10LL) % 1000000LL);
	tv->tv_sec = (long long int) ((now.ns100 - 116444736000000000LL) / 10000000LL);
	
	return (0);
}

double cpuTimer()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void initialData(float *arr, const int size)
{
    time_t t;
    
    srand((unsigned)time(&t));
    
    for (int i=0;i<size;i++)
        arr[i] = (float)(rand())/RAND_MAX;
}

void MatMulOnCPU(float *A, float *B, float *C, 
                    const int Arows, const int Acols, const int Bcols)
{
    float sum;
    
    for (int i=0;i<Arows;i++) {
        for (int j=0;j<Bcols;j++) {
            sum = 0.0f;
            for (int k=0;k<Acols;k++) {
                sum += A[i*Acols+k]*B[k*Bcols+j];
            }
            C[i*Bcols+j] = sum;
        }
    }
}

__global__ void MatMultOnGPU(float *A, float *B, float *C, 
                                const int Arows, const int Acols, const int Bcols)
{
    int tx = blockDim.x*blockIdx.x + threadIdx.x;   // col of C
    int ty = blockDim.y*blockIdx.y + threadIdx.y;   // row of C
    int tid = ty*Bcols+tx;

    float sum=0.0f;

    if (tx<Bcols && ty<Arows) {
        for (int i=0;i<Acols;i++) {
            sum += A[ty*Acols + i]*B[i*Bcols+tx];
        }
        C[tid] = sum;
    }
}

void checkResult(float *host, float *gpu, const int N)
{
    float epsilon = 1.0e-8;
    bool match = 1;

    for (int i=0;i<N;i++) {
        if (abs(host[i]-gpu[i]) > epsilon) {
            match = 0;
            printf("Matrices do not match!\n");
            printf("host %10.7f, gpu %10.7f at current %d\n", host[i], gpu[i], i);
            break;
        }
    }

    if (match) printf("Matrices match.\n");
}

int main(int argc, char **argv)
{
    double Start, ElapsedTime;
    float ETime;
    float *MatA, *MatB, *MatC, *gpu_MatC;
    int Arows=300, Acols=200, Bcols=400;
    int threads_x=32, threads_y=32;

    if(argc>1) Arows=atoi(argv[1]);
    if(argc>2) Acols=atoi(argv[2]);
    if(argc>3) Bcols=atoi(argv[3]);
    if(argc>4) threads_x = atoi(argv[4]);
    if(argc>5) threads_y = atoi(argv[5]);

    /************ ON CPU **************/
    MatA=(float*)malloc(Arows*Acols*sizeof(float));
    MatB=(float*)malloc(Acols*Bcols*sizeof(float));

    initialData(MatA, Arows*Acols);
    initialData(MatB, Acols*Bcols);

    Start=cpuTimer();
    MatC=(float*)malloc(Arows*Bcols*sizeof(float));
    MatMulOnCPU(MatA, MatB, MatC, Arows, Acols, Bcols);
    ElapsedTime=cpuTimer()-Start;
    printf("Elapsed Time on CPU : %f sec\n",ElapsedTime);
    /**********************************/

    /************ ON GPU **************/
    float *d_MatA, *d_MatB, *d_MatC;

    // create two events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    CHECK(hipMalloc((float**)&d_MatA, Arows*Acols*sizeof(float)));
    CHECK(hipMalloc((float**)&d_MatB, Acols*Bcols*sizeof(float)));
    CHECK(hipMalloc((float**)&d_MatC, Arows*Bcols*sizeof(float)));
    CHECK(hipMemcpy(d_MatA,MatA, Arows*Acols*sizeof(float),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_MatB,MatB, Acols*Bcols*sizeof(float),hipMemcpyHostToDevice));
    dim3 block(threads_x,threads_y,1);
    dim3 grid((Bcols+block.x-1)/block.x, (Arows+block.y-1)/block.y, 1);
    MatMultOnGPU<<<grid, block>>>(d_MatA, d_MatB, d_MatC, Arows, Acols, Bcols);
    gpu_MatC=(float*)malloc(Arows*Bcols*sizeof(float));
    CHECK(hipMemcpy(gpu_MatC, d_MatC, Arows*Bcols*sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ETime, start, stop);
    printf("Elapsed Time on GPU : %f sec\n",ETime*1e-3);
    /**********************************/
    //checkResult(MatC, gpu_MatC, Arows*Bcols);
    free(MatA),free(MatB),free(MatC),free(gpu_MatC);
    CHECK(hipFree(d_MatA)), CHECK(hipFree(d_MatB)), CHECK(hipFree(d_MatC));

    CHECK(hipDeviceReset());
    return 0;
}