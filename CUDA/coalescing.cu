#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BlockSize   16  // Size of a block, 16 x 16 threads

__global__ void gpu_Comput (int *h, int N, int T) {
    // Array loaded with global thread ID that acesses that location
	int col = threadIdx.x + blockDim.x * blockIdx.x; 
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	int threadID = col + row * N;
	int index = row + col * N;		// sequentially down each row

	for (int t = 0; t < T; t++)		// loop to repeat to reduce other time effects
	   h[index] = threadID;  		// load array with flattened global thread ID
}

void printArray(int *h, int N) {
	printf("Results of computation, every N/8 numbers, eight numbers\n");

	for (int row = 0; row < N; row += N/8) {
	  for (int col = 0; col < N; col += N/8) 
	 	printf("%6d  ", h[col + row * N]);
	  printf("\n"); 
	}
}

int main(int argc, char *argv[])  {
	int     T = 100;    // number of iterations
	int     B = 1;      // number of blocks
	char    key;
	int     *h, *dev_h; // ptr to array holding numbers on host and device
    float   elapsed_time_ms1;

	hipEvent_t start, stop;    // cuda events to measure time
	
	hipEventCreate( &start ); 
	hipEventCreate( &stop );

    do {  // loop to repeat complete program
        /* ------------------------- Keyboard input -----------------------------------*/
        
        printf("Grid Structure 2-D grid, 2-D blocks\n");
        printf("Blocks fixed at 16 x 16 threads, 512 threads, max for compute cap. 1.x\n");
        printf("Enter number of blocks in grid, each dimension, currently %d\n",B);
        scanf("%d",&B);
        printf("Enter number of iterations, currently %d\n",T);
        scanf("%d",&T);

	    int N = B * BlockSize;  // size of data array, given input data

	    printf("Array size (and total grid-block size) %d x %d\n", N, N);

        dim3    Block(BlockSize, BlockSize);    //Block structure, 32 x 32 max
        dim3    Grid(B, B);                     //Grid structure, B x B	

        /* ------------------------- Allocate Memory-----------------------------------*/

	    int size = N * N * sizeof(int);		// number of bytes in total in array
                                            // B*B*(Blocksize*Blocksize) 
                                            // => num_of_blocks*num_of_threads_per_each_block
        h = (int*) malloc(size);			// Array on host
	    hipMalloc((void**)&dev_h, size);   // allocate device memory

        /* ------------------------- GPU Computation -----------------------------------*/
			
	    hipEventRecord( start, 0 );
	
	    gpu_Comput<<< Grid, Block >>>(dev_h, N, T);
	
        hipEventRecord( stop, 0 );     // instrument code to measue end time
        hipEventSynchronize( stop );   // wait for all work done by threads
        hipEventElapsedTime( &elapsed_time_ms1, start, stop );

	    hipMemcpy(h,dev_h, size ,hipMemcpyDeviceToHost);  //Get results to check

	    printArray(h,N);
	    printf("\nTime to calculate results on GPU: %f ms.\n", elapsed_time_ms1);

        /* -------------------------REPEAT PROGRAM INPUT-----------------------------------*/

	    printf("\nEnter c to repeat, return to terminate\n");

	    scanf("%c",&key);
	    scanf("%c",&key);

    } while (key == 'c'); // loop of complete program
 
    /* --------------  clean up  ---------------------------------------*/
	
	free(h);
	hipFree(dev_h);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
  
	return 0;
}