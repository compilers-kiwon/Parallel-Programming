
#include <hip/hip_runtime.h>
#include    <stdio.h>
#include    <stdlib.h>
#include    <time.h>

#define MAX_SIZE    4000000

#define NUM_OF_BLOCKS               200
#define NUM_OF_THREADS_PER_BLOCK    20000
#define VECTOR_SIZE_PER_THREAD      (MAX_SIZE/(NUM_OF_BLOCKS*NUM_OF_THREADS_PER_BLOCK))

#define DST     0
#define SRC1    1
#define SRC2    2
#define MAX_MEM 3

#define min(a,b)    ((a)<(b)?(a):(b))

__global__ void vector_add(int *dst,int *src1,int* src2)
{
    int begin = (blockDim.x*blockIdx.x+threadIdx.x)*VECTOR_SIZE_PER_THREAD;
    int end = min(begin+VECTOR_SIZE_PER_THREAD,MAX_SIZE);

    for(;begin<end;begin++)
    {
        dst[begin] = src1[begin]+src2[begin];
    }
}

int main(void)
{
    int i;
    int *cpu_mem[MAX_MEM],*cuda_mem[MAX_MEM];

    clock_t start = clock();

    for(i=0;i<MAX_MEM;i++)
    {
        cpu_mem[i] = (int*)malloc(sizeof(int)*MAX_SIZE);
        hipMalloc((void**)&cuda_mem[i],sizeof(int)*MAX_SIZE);
    }

    srand(time(NULL));

    for(i=0;i<MAX_SIZE;i++)
    {
        cpu_mem[SRC1][i] = rand();
        cpu_mem[SRC2][i] = rand();
    }

    hipMemcpy(cuda_mem[SRC1],cpu_mem[SRC1],sizeof(int)*MAX_SIZE,hipMemcpyHostToDevice);
    hipMemcpy(cuda_mem[SRC2],cpu_mem[SRC2],sizeof(int)*MAX_SIZE,hipMemcpyHostToDevice);

    vector_add<<<NUM_OF_BLOCKS,NUM_OF_THREADS_PER_BLOCK>>>(cuda_mem[DST],cuda_mem[SRC1],cuda_mem[SRC2]);
    hipDeviceSynchronize();
    hipMemcpy(cpu_mem[DST],cuda_mem[DST],sizeof(int)*MAX_SIZE,hipMemcpyDeviceToHost);

    for(i=0;i<MAX_MEM;i++)
    {
        free(cpu_mem[i]);
        hipFree(cuda_mem[i]);
    }

    printf("time : %d ms\n",((int)clock() - start) / (CLOCKS_PER_SEC / 1000));
    return  0;
}