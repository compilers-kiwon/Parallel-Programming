
#include <hip/hip_runtime.h>
#include    <stdio.h>
#include    <stdlib.h>
#include    <time.h>

#define MAX_SIZE    4000000

#define NUM_OF_BLOCKS               400
#define NUM_OF_THREADS_PER_BLOCK    1000
#define VECTOR_SIZE_PER_THREAD      (MAX_SIZE/(NUM_OF_BLOCKS*NUM_OF_THREADS_PER_BLOCK))

#define DST     0
#define SRC1    1
#define SRC2    2
#define MAX_MEM 3

#define mem
__global__ void vector_add(int *dst,int *src1,int* src2)
{
    int begin = (blockDim.x*blockIdx.x+threadIdx.x)*VECTOR_SIZE_PER_THREAD;
    int end = begin+VECTOR_SIZE_PER_THREAD;

    //printf("(%d,%d) thread takes vector[%d~%d]\n",
    //                    blockIdx.x,threadIdx.x,begin,end);

    for(;begin<end;begin++)
    {
        dst[begin] = src1[begin]+src2[begin];
    }
}

int main(void)
{
    int i;
    int *cpu_mem[MAX_MEM],*cuda_mem[MAX_MEM];

    clock_t start = clock();

    for(i=0;i<MAX_MEM;i++)
    {
        cpu_mem[i] = (int*)malloc(sizeof(int)*MAX_SIZE);
        hipMalloc((void**)&cuda_mem[i],sizeof(int)*MAX_SIZE);
    }

    srand(time(NULL));

    for(i=0;i<MAX_SIZE;i++)
    {
        cpu_mem[SRC1][i] = rand();
        cpu_mem[SRC2][i] = rand();
    }

    hipMemcpy(cuda_mem[SRC1],cpu_mem[SRC1],sizeof(int)*MAX_SIZE,hipMemcpyHostToDevice);
    hipMemcpy(cuda_mem[SRC2],cpu_mem[SRC2],sizeof(int)*MAX_SIZE,hipMemcpyHostToDevice);

    vector_add<<<NUM_OF_BLOCKS,NUM_OF_THREADS_PER_BLOCK>>>(cuda_mem[DST],cuda_mem[SRC1],cuda_mem[SRC2]);
    hipDeviceSynchronize();
    hipMemcpy(cpu_mem[DST],cuda_mem[DST],sizeof(int)*MAX_SIZE,hipMemcpyDeviceToHost);
/*
    for(i=0;i<MAX_SIZE;i++)
    {
        if( cpu_mem[SRC1][i]+cpu_mem[SRC2][i] != cpu_mem[DST][i] )
        {
            printf("Unexpected!!\n");
            printf("[%07d] %d = %d + %d\n",i,cpu_mem[DST][i],cpu_mem[SRC1][i],cpu_mem[SRC2][i]);
            break;
        }
        else
        {
            if( i%1000000 == 0 )
            {
                printf("[%07d] %d = %d + %d\n",i,cpu_mem[DST][i],cpu_mem[SRC1][i],cpu_mem[SRC2][i]);
            }
        }
    }
*/
    for(i=0;i<MAX_MEM;i++)
    {
        free(cpu_mem[i]);
        hipFree(cuda_mem[i]);
    }

    printf("time : %d ms\n",((int)clock() - start) / (CLOCKS_PER_SEC / 1000));
    return  0;
}